#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hipblas.h>
#include <float.h>
#include <stdio.h>

__global__ void __kernel__multiply_array(float* array, float value, int num_kernels){
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    for (; index < num_kernels; index += blockDim.x*gridDim.x){
        array[index] *= value;
    }
}

void __cuda__multiply_array(float* array, const float value, const size_t size, const bool allocated){

    float * cuda_array=nullptr;
    int blockSize=512;

    if(!allocated){
        hipMalloc(&cuda_array, size * sizeof(float));
        hipMemcpy(cuda_array, array, size * sizeof(float), hipMemcpyHostToDevice);
    }
    else{
        cuda_array=array;
    }

    __kernel__multiply_array <<<(size + blockSize - 1) / blockSize,
  	  blockSize, 0/*, get_cuda_stream() */>>>
  		(cuda_array, value, size);

    hipDeviceSynchronize();

    if(!allocated){
        hipMemcpy(array, cuda_array, size * sizeof(float), hipMemcpyDeviceToHost);
    }
}
